//
// Created by 张晓彬 on 2021/9/5.
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


void create(int size)
{
    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i)
        hipStreamCreate(&stream[i]);
    float* hostptr;
    hipHostMalloc(&hostptr, 2*size, hipHostMallocDefault);
}