//
// Created by 张晓彬 on 2021/9/4.
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 16


typedef struct
{
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;


__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row*A.stride+col];
}


__device__ void SetElement(Matrix A, int row, int col, float val)
{
    A.elements[row*A.stride+col] = val;
}


__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width = BLOCK_SIZE;
    Asub.height = BLOCK_SIZE;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride*BLOCK_SIZE*row+BLOCK_SIZE*col];
    return Asub;
}


__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);


void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    Matrix d_A;
    d_A.width = d_A.stride = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = B.width;
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    Matrix d_C;
    d_C.width = d_C.stride = C.width;
    d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width/dimBlock.x, A.height/dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}


__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
    float cval = 0;
    int row = threadIdx.y;
    int col = threadIdx.x;
    for (int m = 0; m < (A.width/BLOCK_SIZE); ++m)
    {
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        Matrix Bsub = GetSubMatrix(B, m, blockCol);
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);
        __syncthreads();
        for (int e = 0; e < BLOCK_SIZE; ++e)
            cval += As[row][e] * Bs[e][col];
        __syncthreads();
    }
    SetElement(Csub, row, col, cval);
}
